
#include <hip/hip_runtime.h>
__device__ void mybarrier() {
  __syncthreads();
}

__global__ void reduce(int *g_idata, int *g_odata) {
  extern __shared__ int sdata[];
  // each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  sdata[tid] = g_idata[i];

  __shared__ float toto;
  toto = 0.0;
  __syncthreads(); // OK

  if (tid == 0) {
    toto = 1.0;
    __syncthreads(); // error
  }


  if (toto)
    __syncthreads(); // error

  mybarrier();

  // Now toto has the same value for all threads in the workgroup.
  if (toto)
    __syncthreads(); // OK

  // do reduction in shared mem
  for(unsigned int s=1; s < blockDim.x; s *= 2) {
    if (tid % (2*s) == 0) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads(); // OK
  }
  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];

  if (tid == 0)
    toto = 1.0;

  __syncthreads();

  if (toto > 10)
    __syncthreads(); // OK
}